
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <algorithm>
#include <iomanip>

typedef double Real;

__global__
void add(int n, Real* x, Real* y){
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(int i=index; i<n; i+=stride){
		y[i] = x[i] + y[i];
	}
}

int main(){
    std::cout << std::fixed << std::setprecision(20);

	int n = 1<<20;

	Real *x, *y;
	hipMallocManaged(&x, n*sizeof(Real));
	hipMallocManaged(&y, n*sizeof(Real));

	for(int i=0; i<n; i++){
		x[i] = static_cast<Real>(1);
		y[i] = static_cast<Real>(2);
	}

	int blocksize = 32*8;
	int numBlock = (n + blocksize - 1) / blocksize;
	
	add<<<numBlock, blocksize>>>(n, x, y);
	hipDeviceSynchronize();

	Real maxError = static_cast<Real>(0);

	for(int i=0; i<n; ++i){
		maxError = std::max(maxError, std::fabs(y[i] - static_cast<Real>(3)));
	}

	std::cout << "Max Error: " << maxError << std::endl;

	hipFree(x);
	hipFree(y);
}
